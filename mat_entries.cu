#include "hip/hip_runtime.h"
#include "mat_entries.h"

__global__
void mass(int *arr_m, double *mass_array,int n)
{
    int i = threadIdx.x;
    int k=blockIdx.x;
    int kk=k*1024;
    
    
    double dd=0;
    int oj,ii;
    if(i+kk<n)
    {
        ii=(i+kk)*28;
        for(oj=0;oj<28;oj=oj+2)
        {
            if(arr_m[ii+oj]!=0)
            {
                double mass_d;
                mass_d=(double) arr_m[ii+oj+1]/arr_m[ii+oj];
                dd=dd+mass_d;

            }
        }
        mass_array[i+kk]=dd;
    }
    //printf("mass=%f\n\n",dd);
} 

__global__
void diagonal(int *arr,double *diag_array,int n,double coupling)
{
    
    int j,ii;
    double a,s,c,v;
    
    int i = threadIdx.x;
    int k=blockIdx.x;
    int kk=k*1024;
    double diag_H=0;
    if(i+kk<n)    
    {
        ii=(i+kk)*28;
        for(j=0;j<28;j=j+2)
        {
            if(arr[ii+j]==0)
            {
                break;
            }
            if(arr[ii+j]!=0)
            {
                double dd1; 

                dd1=(double) (arr[ii+j+1]*(arr[ii+j+1]-1))/(arr[ii+j]*arr[ii+j]);
                diag_H=diag_H+ ((coupling*dd1)/(16*M_PI));
           
                for(k=j+2;k<28;k=k+2)
                {
                    if(arr[ii+k]!=0)
                    {
                        a=arr[ii+j];
                        s=arr[ii+k];
                        c=arr[ii+j+1];
                        v=arr[ii+k+1];
                        diag_H=diag_H+((4*coupling*c*v)/(a*s*16*M_PI));
                    }
                }

            }
        }
        diag_array[i+kk]=diag_H; 
    }
   // printf("diag_H=%f\n\n",diag_H);
   // return diag_H;
}
