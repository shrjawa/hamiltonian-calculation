#include "hip/hip_runtime.h"

#include <math.h>
#include<stdio.h>
#include<time.h>
#include "mat_entries.h"
#define IDX(kk,ll) kk*ll



__global__
void partition(int m,int *pre_array)
{
    int j;
    int c=0;
    int *p=(int *)malloc(m);
    int ii=0;
    int  k = 0;
    p[k] = m;
    int temp;
    while (1)
    {
        
        if(k%2==0)
        {
            for(j=0;j<k+1;j++)
            {
                pre_array[j+c]=p[j];
            }
            ii++;
            c+=m;
        }
        temp = 0;
        while (k >= 0 && p[k] == 1)
        {
            temp += 1;
            k--;
        }

        if (k < 0)  break;

        p[k]--;
        temp++;

        while (temp > p[k])
        {
            p[k+1] = p[k];
            temp = temp - p[k];
            k++;
        }

        p[k+1] = temp;
        k++;
        
    }
    
}



__global__
void array_create(int *pre_array,int *main_array,int n,int momentum)
{
   
    int i;
    int j,k;
    int mai=0;
    
    for(i=0;i<n*momentum;i=i+momentum)
    {
        
        int kk=0;
        
        int previous_num=0;
        for(j=0;j<momentum;j++)
        {
            if(pre_array[i+j]!=0)
            {
                if(pre_array[i+j]==previous_num)
                {
                    main_array[mai+kk-1]+=1;
                }
                if(pre_array[i+j]!=previous_num)
                {
                    int appendornot=0;
                    int number=pre_array[i+j];
                    for(k=0;k<28;k=k+2)
                    {
                        if(number==main_array[mai+k])
                        {
                            appendornot+=1;
                        }
                        if(main_array[mai+k]==0)
                        {
                            break;
                        }
                    }
                    
                    if(appendornot==0)
                    {    
                        
                        main_array[mai+kk]=number;
                        main_array[mai+kk+1]=1;
                        kk+=2;
                    }
                    previous_num=pre_array[i+j];
                }
            }
            
        } 
       
        mai=mai+28;
        
    }
}



int main(void)
{
    clock_t start = clock();
    int momentum=16;
    int n=113;  //dimension
    double coupling=1;
    int *pre_array;
    
    hipMallocManaged(&pre_array,IDX(n,momentum)*sizeof(int)); //array initialization of partition of integer momentum
    
    for(int i=0;i<IDX(n,momentum);i++)
    {
	    pre_array[i]=0;
    }
    partition<<<1,1>>>(momentum,pre_array);
    hipDeviceSynchronize();
    
   /*for(int i=0;i<IDX(n,momentum);i++)
    {
           printf("%i,",pre_array[i]);
    } */
    printf("\n");
    
    //main_array of making proper state 
    int *main_array;
    hipMallocManaged(&main_array,IDX(n,28)*sizeof(int));   //every 28 places new state
    
    for(int i=0;i<IDX(n,28);i++)
    {
        main_array[i]=0; 
    }
    array_create<<<1,1>>>(pre_array ,main_array,n,momentum);
    hipDeviceSynchronize();
   
    /*for(int i=0;i<n*28;i++)
    {
        
        if(main_array[i]!=0)
        {
            printf("%i,",main_array[i]);
        }
        
    }*/
  
    
    hipFree(pre_array);//remove pre_array as we are done with it 
    
    double ii=(double) n/1024;
    printf("ii=%i\n",ii);
    ii=ceil(ii);
    printf("ii=%i\n",ii);
    double *mass_array;
    hipMallocManaged(&mass_array,n*sizeof(double));
    mass<<<ii ,1024>>>(main_array,mass_array,n);
    hipDeviceSynchronize();
    /*printf("mass elements");
    for(int i =0;i<n;i++)
    {
        printf("%f,",mass_array[i]);
    }*/
    double *diag_array;
    hipMallocManaged(&diag_array,n*sizeof(double));
    diagonal<<<ii ,1024>>>(main_array,diag_array,n,coupling);
    hipDeviceSynchronize();
    printf("diag elements");
    for(int i =0;i<n;i++)
    {
        printf("%f,",momentum*(mass_array[i]+diag_array[i]));
    }
    clock_t end = clock();
    printf("\ntime taken=%d",end - start);
    printf("\n");
    
     
    hipFree(main_array);
    
    hipFree(mass_array);
    
    
}
