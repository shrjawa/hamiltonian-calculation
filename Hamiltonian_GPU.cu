#include "hip/hip_runtime.h"

#include <math.h>
#include<stdio.h>
#include<time.h>
#include "mat_entries.h"
#include "states_creation.h"

#define IDX(kk,ll) kk*ll




int main(void)
{
    clock_t start = clock();
    int momentum=16;
    int n=113;  //dimension
    double coupling=1;
    int *pre_array;
    
    hipMallocManaged(&pre_array,IDX(n,momentum)*sizeof(int)); //array initialization of partition of integer momentum
    
    for(int i=0;i<IDX(n,momentum);i++)
    {
	    pre_array[i]=0;
    }
    partition<<<1,1>>>(momentum,pre_array);
    hipDeviceSynchronize();
    
   /*for(int i=0;i<IDX(n,momentum);i++)
    {
           printf("%i,",pre_array[i]);
    } */
    printf("\n");
    
    //main_array of making proper state 
    int *main_array;
    hipMallocManaged(&main_array,IDX(n,28)*sizeof(int));   //every 28 places new state
    
    for(int i=0;i<IDX(n,28);i++)
    {
        main_array[i]=0; 
    }
    array_create<<<1,1>>>(pre_array ,main_array,n,momentum);
    hipDeviceSynchronize();
   
    /*for(int i=0;i<n*28;i++)
    {
        
        if(main_array[i]!=0)
        {
            printf("%i,",main_array[i]);
        }
        
    }*/
  
    
    hipFree(pre_array);//remove pre_array as we are done with it 
    
    double ii=(double) n/1024;
    printf("ii=%i\n",ii);
    ii=ceil(ii);
    printf("ii=%i\n",ii);
    double *mass_array;
    hipMallocManaged(&mass_array,n*sizeof(double));
    mass<<<ii ,1024>>>(main_array,mass_array,n);
    hipDeviceSynchronize();
    /*printf("mass elements");
    for(int i =0;i<n;i++)
    {
        printf("%f,",mass_array[i]);
    }*/
    double *diag_array;
    hipMallocManaged(&diag_array,n*sizeof(double));
    diagonal<<<ii ,1024>>>(main_array,diag_array,n,coupling);
    hipDeviceSynchronize();
    printf("diag elements");
    for(int i =0;i<n;i++)
    {
        printf("%f,",momentum*(mass_array[i]+diag_array[i]));
    }
    clock_t end = clock();
    printf("\ntime taken=%d",end - start);
    printf("\n");
    
     
    hipFree(main_array);
    
    hipFree(mass_array);
    
    
}
