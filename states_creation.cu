#include "hip/hip_runtime.h"
#include "states_creation.h"

__global__
void partition(int m,int *pre_array)
{
    int j;
    int c=0;
    int *p=(int *)malloc(m);
    int ii=0;
    int  k = 0;
    p[k] = m;
    int temp;
    while (1)
    {
        
        if(k%2==0)
        {
            for(j=0;j<k+1;j++)
            {
                pre_array[j+c]=p[j];
            }
            ii++;
            c+=m;
        }
        temp = 0;
        while (k >= 0 && p[k] == 1)
        {
            temp += 1;
            k--;
        }

        if (k < 0)  break;

        p[k]--;
        temp++;

        while (temp > p[k])
        {
            p[k+1] = p[k];
            temp = temp - p[k];
            k++;
        }

        p[k+1] = temp;
        k++;
        
    }
    
}



__global__
void array_create(int *pre_array,int *main_array,int n,int momentum)
{
   
    int i;
    int j,k;
    int mai=0;
    
    for(i=0;i<n*momentum;i=i+momentum)
    {
        
        int kk=0;
        
        int previous_num=0;
        for(j=0;j<momentum;j++)
        {
            if(pre_array[i+j]!=0)
            {
                if(pre_array[i+j]==previous_num)
                {
                    main_array[mai+kk-1]+=1;
                }
                if(pre_array[i+j]!=previous_num)
                {
                    int appendornot=0;
                    int number=pre_array[i+j];
                    for(k=0;k<28;k=k+2)
                    {
                        if(number==main_array[mai+k])
                        {
                            appendornot+=1;
                        }
                        if(main_array[mai+k]==0)
                        {
                            break;
                        }
                    }
                    
                    if(appendornot==0)
                    {    
                        
                        main_array[mai+kk]=number;
                        main_array[mai+kk+1]=1;
                        kk+=2;
                    }
                    previous_num=pre_array[i+j];
                }
            }
            
        } 
       
        mai=mai+28;
        
    }
}


